#include "hip/hip_runtime.h"
#include <stdio.h>
#include "Types.H"
#include "hipblas.h"
#include "hipsparse.h"
#include "hip/hip_runtime.h"

#ifndef max_stream
#define max_stream 16
#endif

#ifndef BLOCK_DIM
#define BLOCK_DIM 16
#endif

static volatile size_t c_memory = 0;

extern "C"
void set_gpu(int dev,char *gpu_string){
     struct hipDeviceProp_t dprop;
     hipSetDevice(dev);
     hipGetDeviceProperties(&dprop, dev);
     strcpy(gpu_string,dprop.name);	
}

extern "C"
void cublas_init(void **handle){
     hipblasCreate((hipblasHandle_t*)handle);
}

extern "C"
void cublas_finalize(void *handle){
     hipblasDestroy((hipblasHandle_t)handle);
}

extern "C"
void cusparse_init(void **handle){
     hipsparseCreate((hipsparseHandle_t*)handle);
}

extern "C"
void cusparse_finalize(void *handle){
     hipsparseDestroy((hipsparseHandle_t)handle);
}

extern "C"
size_t allocate_data_on_device(void **data,size_t size_data){

     hipError_t mem_error;  

     mem_error = hipMalloc(data,size_data);

     if(mem_error!=hipSuccess){
         printf("CPU wants to allocate %e MBytes on the device, but already %e MBytes are in use\n",size_data/1e6,c_memory/1e6);
	 exit(0);
     }else{
         c_memory = c_memory+size_data;
     }

     return c_memory;
}

extern "C"
void deallocate_data_on_device(void *data){
     hipFree(data);
}

extern "C"
size_t deallocate_data_on_dev(void *data,size_t size_data){

     hipFree(data);

     c_memory = c_memory-size_data;

     return c_memory;
}

extern "C"
void copy_data_to_device(void *host_data,void *device_data,int N,int M,size_t size_element){
     hipblasSetMatrixAsync(N,M,size_element,host_data,N,device_data,N,NULL);
}

extern "C"
void memcpy_to_device(void *host_data,void *device_data,size_t size_element){
     hipMemcpyAsync(device_data,host_data,size_element,hipMemcpyHostToDevice,NULL);
}

extern "C"
void copy_data_to_host(void *host_data,void *device_data,int N,int M,size_t size_element){
     hipblasGetMatrixAsync(N,M,size_element,device_data,N,host_data,N,NULL);
}

extern "C"
void memcpy_to_host(void *host_data,void *device_data,size_t size_element){
     hipMemcpyAsync(host_data,device_data,size_element,hipMemcpyDeviceToHost,NULL);
}

extern "C"
void dgemm_on_dev(void *handle,char transa,char transb,int m,int n,int k,double alpha,\
      		  double *A,int lda,double *B,int ldb,double beta,double *C, int ldc){
     
     hipblasOperation_t opA,opB;

     if(transa=='N'){
	opA = HIPBLAS_OP_N;
     }
     if(transa=='C'){
	opA = HIPBLAS_OP_C;
     }
     if(transa=='T'){
	opA = HIPBLAS_OP_T;
     }

     if(transb=='N'){
	opB = HIPBLAS_OP_N;
     }
     if(transb=='C'){
	opB = HIPBLAS_OP_C;
     }
     if(transb=='T'){
	opB = HIPBLAS_OP_T;
     }

     hipblasDgemm((hipblasHandle_t)handle,opA,opB,m,n,k,&alpha,A,lda,B,ldb,&beta,C,ldc);
}

extern "C"
void zgemm_on_dev(void *handle,char transa,char transb,int m,int n,int k,CPX alpha,\
                  CPX *A,int lda,CPX *B,int ldb,CPX beta,CPX *C, int ldc){
  
     hipblasOperation_t opA,opB;

     if(transa=='N'){
	opA = HIPBLAS_OP_N;
     }
     if(transa=='C'){
	opA = HIPBLAS_OP_C;
     }
     if(transa=='T'){
	opA = HIPBLAS_OP_T;
     }

     if(transb=='N'){
	opB = HIPBLAS_OP_N;
     }
     if(transb=='C'){
	opB = HIPBLAS_OP_C;
     }
     if(transb=='T'){
	opB = HIPBLAS_OP_T;
     }

     hipblasZgemm((hipblasHandle_t)handle,opA,opB,m,n,k,(hipDoubleComplex*)&alpha,\
                 (hipDoubleComplex*)A,lda,(hipDoubleComplex*)B,ldb,(hipDoubleComplex*)&beta,\
		 (hipDoubleComplex*)C,ldc);
}

extern "C"
void zaxpy_on_dev(void *handle,int n,CPX alpha,CPX *x,int incx,CPX *y,int incy){
  
    hipblasZaxpy((hipblasHandle_t)handle,n,(hipDoubleComplex*)&alpha,(hipDoubleComplex*)x,\
                incx,(hipDoubleComplex*)y,incy);
}

__global__ void d_init_variable_on_dev(double *var,int N){

     int idx = blockIdx.x*blockDim.x + threadIdx.x;

     if(idx<N){
	var[idx] = 0.0;
     }	   

     __syncthreads();
}

extern "C"
void d_init_var_on_dev(double *var,int N,hipStream_t stream){

    uint i_N = N + (BLOCK_DIM-(N%BLOCK_DIM));

    d_init_variable_on_dev<<< i_N/BLOCK_DIM, BLOCK_DIM, 0, stream >>>(var,N);
}

__global__ void d_init_eye_on_device(double *var,int N){

     int idx = blockIdx.x*blockDim.x + threadIdx.x;

     if(idx<N*N){
	var[idx] = 0.0;
	if(!(idx%(N+1))){
	    var[idx] = 1.0;
	}	   
     }

     __syncthreads();
}

extern "C"
void d_init_eye_on_dev(double *var,int N,hipStream_t stream){

    uint i_N = N*N + (BLOCK_DIM-((N*N)%BLOCK_DIM));

    d_init_eye_on_device<<< i_N/BLOCK_DIM, BLOCK_DIM, 0, stream >>>(var,N);
}

__global__ void z_init_variable_on_dev(hipDoubleComplex *var,int N){

     int idx = blockIdx.x*blockDim.x + threadIdx.x;

     if(idx<N){
	var[idx].x = 0.0;
	var[idx].y = 0.0;
     }	   

     __syncthreads();
}

extern "C"
void z_init_var_on_dev(CPX *var,int N,hipStream_t stream){

    uint i_N = N + (BLOCK_DIM-(N%BLOCK_DIM));

    z_init_variable_on_dev<<< i_N/BLOCK_DIM, BLOCK_DIM, 0, stream >>>((hipDoubleComplex*)var,N);
}

__global__ void z_init_eye_on_device(hipDoubleComplex *var,int N){

     int idx = blockIdx.x*blockDim.x + threadIdx.x;

     if(idx<N*N){
	var[idx].x = 0.0;
	var[idx].y = 0.0;
	if(!(idx%(N+1))){
	    var[idx].x = 1.0;
	}	   
     }

     __syncthreads();
}

extern "C"
void z_init_eye_on_dev(CPX *var,int N,hipStream_t stream){

    uint i_N = N*N + (BLOCK_DIM-((N*N)%BLOCK_DIM));

    z_init_eye_on_device<<< i_N/BLOCK_DIM, BLOCK_DIM, 0, stream >>>((hipDoubleComplex*)var,N);
}

__global__ void correct_diag_on_device(hipDoubleComplex *var,int N){

     int idx = blockIdx.x*blockDim.x + threadIdx.x;

     if((idx<N*N)&&(!(idx%(N+1)))){
         var[idx].y = 0.0;	   
     }

     __syncthreads();
}

extern "C"
void correct_diag_on_dev(CPX *var,int N,hipStream_t stream){

    uint i_N = N*N + (BLOCK_DIM-((N*N)%BLOCK_DIM));

    correct_diag_on_device<<< i_N/BLOCK_DIM, BLOCK_DIM, 0, stream >>>((hipDoubleComplex*)var,N);
}

__global__ void change_variable_type_on_dev(double *var1,hipDoubleComplex *var2,int N){

     int idx = blockIdx.x*blockDim.x + threadIdx.x;

     if(idx<N){
	var2[idx].x = var1[idx];
	var2[idx].y = 0.0;
     }	   

     __syncthreads();
}

extern "C"
void change_var_type_on_dev(double *var1,CPX *var2,int N,hipStream_t stream){

    uint i_N = N + (BLOCK_DIM-(N%BLOCK_DIM));

    change_variable_type_on_dev<<< i_N/BLOCK_DIM, BLOCK_DIM, 0, stream >>>(var1,(hipDoubleComplex*)var2,N);
}

__global__ void change_sign_imaginary_part_on_dev(hipDoubleComplex *var,int N){

     int idx = blockIdx.x*blockDim.x + threadIdx.x;

     if(idx<N){
	var[idx].y = -var[idx].y;
     }	   

     __syncthreads();
}

extern "C"
void change_sign_imag_on_dev(CPX *var,int N){

    uint i_N = N + (BLOCK_DIM-(N%BLOCK_DIM));

    change_sign_imaginary_part_on_dev<<< i_N/BLOCK_DIM, BLOCK_DIM >>>((hipDoubleComplex*)var,N);
}

__global__ void d_extract_diag(double *D,int *edge_i,int *index_j,double *nnz,\
	   int NR,int imin,int imax,int shift,int findx){

     int j;
     int ind_j;	   
     int idx = blockIdx.x*blockDim.x + threadIdx.x;

     if(idx<NR){
	  for(j=edge_i[idx+imin]-findx;j<edge_i[idx+imin+1]-findx;j++){
	      ind_j = index_j[j]-findx-shift-imin;
	      if((ind_j>=0)&&(ind_j<NR)){
	          D[idx+ind_j*NR] = nnz[j];
	      }
	  }
     }	   

     __syncthreads();
}

extern "C"
void d_extract_diag_on_dev(double *D,int *edge_i,int *index_j,double *nnz,int NR,\
     int imin,int imax,int shift,int findx,hipStream_t stream){

    uint i_N = NR + (BLOCK_DIM-(NR%BLOCK_DIM));

    d_extract_diag<<< i_N/BLOCK_DIM, BLOCK_DIM, 0, stream >>>(D,edge_i,index_j,nnz,NR,imin,imax,shift,findx);
}

__global__ void d_extract_not_diag(double *D,int *edge_i,int *index_j,double *nnz,\
	   int NR,int imin,int imax,int jmin,int side,int shift,int findx){

     int j;
     int ind_j;	   
     int limit = 0;
     int idx   = blockIdx.x*blockDim.x + threadIdx.x;

     if(side==-1){
         limit = -(imin+shift-jmin-1);
     }

     if(idx<NR){
	  for(j=edge_i[idx+imin]-findx;j<edge_i[idx+imin+1]-findx;j++){
	      ind_j = index_j[j]-findx-jmin;
	      if(side*ind_j>=limit){
	          D[idx+ind_j*NR] = nnz[j];
	      }
	  }
     }	   

     __syncthreads();
}

extern "C"
void d_extract_not_diag_on_dev(double *D,int *edge_i,int *index_j,double *nnz,int NR,\
     int imin,int imax,int jmin,int side,int shift,int findx,hipStream_t stream){

    uint i_N = NR + (BLOCK_DIM-(NR%BLOCK_DIM));

    d_extract_not_diag<<< i_N/BLOCK_DIM, BLOCK_DIM, 0, stream >>>(D,edge_i,index_j,nnz,NR,imin,imax,jmin,side,shift,findx);
}

__global__ void z_extract_diag(hipDoubleComplex *D,int *edge_i,int *index_j,hipDoubleComplex *nnz,\
	   int NR,int imin,int imax,int shift,int findx){

     int j;
     int ind_j;	   
     int idx = blockIdx.x*blockDim.x + threadIdx.x;

     if(idx<NR){
	  for(j=edge_i[idx+imin]-findx;j<edge_i[idx+imin+1]-findx;j++){
	      ind_j = index_j[j]-findx-shift-imin;
	      if((ind_j>=0)&&(ind_j<NR)){
	          D[idx+ind_j*NR].x = nnz[j].x;
		  D[idx+ind_j*NR].y = nnz[j].y;
	      }
	  }
     }	   

     __syncthreads();
}

extern "C"
void z_extract_diag_on_dev(CPX *D,int *edge_i,int *index_j,CPX *nnz,int NR,\
     int imin,int imax,int shift,int findx,hipStream_t stream){

    uint i_N = NR + (BLOCK_DIM-(NR%BLOCK_DIM));

    z_extract_diag<<< i_N/BLOCK_DIM, BLOCK_DIM, 0, stream >>>((hipDoubleComplex*)D,edge_i,index_j,(hipDoubleComplex*)nnz,NR,imin,imax,shift,findx);
}

__global__ void z_extract_not_diag(hipDoubleComplex *D,int *edge_i,int *index_j,hipDoubleComplex *nnz,\
	   int NR,int imin,int imax,int jmin,int side,int shift,int findx){

     int j;
     int ind_j;	   
     int limit = 0;
     int idx   = blockIdx.x*blockDim.x + threadIdx.x;

     if(side==-1){
         limit = -(imin+shift-jmin-1);
     }

     if(idx<NR){
	  for(j=edge_i[idx+imin]-findx;j<edge_i[idx+imin+1]-findx;j++){
	      ind_j = index_j[j]-findx-jmin;
	      if(side*ind_j>=limit){
	          D[idx+ind_j*NR].x = nnz[j].x;
		  D[idx+ind_j*NR].y = nnz[j].y;
	      }
	  }
     }	   

     __syncthreads();
}

extern "C"
void z_extract_not_diag_on_dev(CPX* *D,int *edge_i,int *index_j,CPX *nnz,int NR,\
     int imin,int imax,int jmin,int side,int shift,int findx,hipStream_t stream){

    uint i_N = NR + (BLOCK_DIM-(NR%BLOCK_DIM));

    z_extract_not_diag<<< i_N/BLOCK_DIM, BLOCK_DIM, 0, stream >>>((hipDoubleComplex*)D,edge_i,index_j,(hipDoubleComplex*)nnz,NR,imin,imax,jmin,side,shift,findx);
}

extern "C"
void d_copy_csr_to_device(int size,int n_nonzeros,int *hedge_i,int *hindex_j,double *hnnz,\
          		  int *dedge_i,int *dindex_j,double *dnnz){
    
    hipMemcpyAsync(dedge_i,hedge_i,(size+1)*sizeof(int),hipMemcpyHostToDevice,NULL);
    hipMemcpyAsync(dindex_j,hindex_j,n_nonzeros*sizeof(int),hipMemcpyHostToDevice,NULL);
    hipMemcpyAsync(dnnz,hnnz,n_nonzeros*sizeof(double),hipMemcpyHostToDevice,NULL);
}

extern "C"
void z_copy_csr_to_device(int size,int n_nonzeros,int *hedge_i,int *hindex_j,CPX *hnnz,\
          		  int *dedge_i,int *dindex_j,CPX *dnnz){
    
    hipMemcpyAsync(dedge_i,hedge_i,(size+1)*sizeof(int),hipMemcpyHostToDevice,NULL);
    hipMemcpyAsync(dindex_j,hindex_j,n_nonzeros*sizeof(int),hipMemcpyHostToDevice,NULL);
    hipMemcpyAsync(dnnz,hnnz,n_nonzeros*sizeof(CPX),hipMemcpyHostToDevice,NULL);
}

extern "C"
void d_csr_mult_f(void *handle,int m,int n,int k,int n_nonzeros,int *Aedge_i,int *Aindex_j,\
                  double *Annz,double alpha,double *B,double beta,double *C){

    hipsparseMatDescr_t descra;

    hipsparseCreateMatDescr(&descra);
    hipsparseSetMatType(descra,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descra,HIPSPARSE_INDEX_BASE_ONE);

    hipsparseDcsrmm((hipsparseHandle_t)handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,m,n,k,n_nonzeros,\
                   &alpha,descra,Annz,Aedge_i,Aindex_j,B,k,&beta,C,m);

    hipsparseDestroyMatDescr(descra);
}

extern "C"
void z_csr_mult_f(void *handle,int m,int n,int k,int n_nonzeros,int *Aedge_i,int *Aindex_j,\
                  CPX *Annz,CPX alpha,CPX *B,CPX beta,CPX *C){

    hipsparseMatDescr_t descra;

    hipsparseCreateMatDescr(&descra);
    hipsparseSetMatType(descra,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descra,HIPSPARSE_INDEX_BASE_ONE);

    hipsparseZcsrmm((hipsparseHandle_t)handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,m,n,k,n_nonzeros,\
                   (hipDoubleComplex*)&alpha,descra,(hipDoubleComplex*)Annz,Aedge_i,Aindex_j,\
		   (hipDoubleComplex*)B,k,(hipDoubleComplex*)&beta,(hipDoubleComplex*)C,m);

    hipsparseDestroyMatDescr(descra);
}

extern "C"
void z_csr_mult_fo(void *handle,int m,int n,int k,int n_nonzeros,int *Aedge_i,int *Aindex_j,\
                   CPX *Annz,CPX alpha,CPX *B,CPX beta,CPX *C){

    hipsparseMatDescr_t descra;

    hipsparseCreateMatDescr(&descra);
    hipsparseSetMatType(descra,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descra,HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseZcsrmm((hipsparseHandle_t)handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,m,n,k,n_nonzeros,\
                   (hipDoubleComplex*)&alpha,descra,(hipDoubleComplex*)Annz,Aedge_i,Aindex_j,\
		   (hipDoubleComplex*)B,k,(hipDoubleComplex*)&beta,(hipDoubleComplex*)C,m);

    hipsparseDestroyMatDescr(descra);
}

// This kernel is optimized to ensure all global reads and writes are coalesced,
// and to avoid bank conflicts in shared memory.  This kernel is up to 11x faster
// than the naive kernel below.  Note that the shared memory array is sized to 
// (BLOCK_DIM+1)*BLOCK_DIM.  This pads each row of the 2D block in shared memory 
// so that bank conflicts do not occur when threads address the array column-wise.
__global__ void d_transpose(double *odata, double *idata, int width, int height)
{
	__shared__ double block[BLOCK_DIM][BLOCK_DIM+1];
	
	// read the matrix tile into shared memory
	unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
	unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;
	if((xIndex < width) && (yIndex < height))
	{
		unsigned int index_in = yIndex * width + xIndex;
		block[threadIdx.y][threadIdx.x] = idata[index_in];
	}

	__syncthreads();

	// write the transposed matrix tile to global memory
	xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
	yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;
	if((xIndex < height) && (yIndex < width))
	{
		unsigned int index_out = yIndex * height + xIndex;
		odata[index_out] = block[threadIdx.x][threadIdx.y];
	}
}

extern "C"
void d_transpose_matrix(double *odata,double *idata,int size_x,int size_y){

    uint i_size_x = size_x + (BLOCK_DIM-(size_x%BLOCK_DIM));
    uint i_size_y = size_y + (BLOCK_DIM-(size_y%BLOCK_DIM));

    dim3 grid(i_size_x / BLOCK_DIM, i_size_y / BLOCK_DIM, 1);
    dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);

    d_transpose<<< grid, threads >>>(odata, idata, size_x, size_y);
}

// This kernel is optimized to ensure all global reads and writes are coalesced,
// and to avoid bank conflicts in shared memory.  This kernel is up to 11x faster
// than the naive kernel below.  Note that the shared memory array is sized to 
// (BLOCK_DIM+1)*BLOCK_DIM.  This pads each row of the 2D block in shared memory 
// so that bank conflicts do not occur when threads address the array column-wise.
__global__ void z_transpose(hipDoubleComplex *odata, hipDoubleComplex *idata, int width, int height)
{
	__shared__ hipDoubleComplex block[BLOCK_DIM][BLOCK_DIM+1];
	
	// read the matrix tile into shared memory
	unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
	unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;
	if((xIndex < width) && (yIndex < height))
	{
		unsigned int index_in = yIndex * width + xIndex;
		block[threadIdx.y][threadIdx.x] = idata[index_in];
	}

	__syncthreads();

	// write the transposed matrix tile to global memory
	xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
	yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;
	if((xIndex < height) && (yIndex < width))
	{
		unsigned int index_out = yIndex * height + xIndex;
		odata[index_out].x = block[threadIdx.x][threadIdx.y].x;
		odata[index_out].y = -block[threadIdx.x][threadIdx.y].y;
	}
}

extern "C"
void z_transpose_matrix(CPX *odata,CPX *idata,int size_x,int size_y){

    uint i_size_x = size_x + (BLOCK_DIM-(size_x%BLOCK_DIM));
    uint i_size_y = size_y + (BLOCK_DIM-(size_y%BLOCK_DIM));

    dim3 grid(i_size_x / BLOCK_DIM, i_size_y / BLOCK_DIM, 1);
    dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);

    z_transpose<<< grid, threads >>>((hipDoubleComplex*)odata,(hipDoubleComplex*)idata,size_x,size_y);
}

__global__ void d_symmetrize(double *matrix, int N)
{

        unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
        unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;

        if((xIndex < N) && (yIndex < N) && (yIndex>=xIndex)){
            unsigned int index_1  = yIndex * N + xIndex;
            unsigned int index_2  = xIndex * N + yIndex;
            double val_1    = matrix[index_1];
            double val_2    = matrix[index_2];

            matrix[index_1] = (val_1+val_2)/2.0;
            matrix[index_2] = (val_1+val_2)/2.0;
        }

        __syncthreads();
}

extern "C"
void d_symmetrize_matrix(double *matrix,int N,hipStream_t stream){

    uint i_size = N + (BLOCK_DIM-(N%BLOCK_DIM));

    dim3 grid(i_size / BLOCK_DIM, i_size / BLOCK_DIM, 1);
    dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);

    d_symmetrize<<< grid, threads, 0, stream >>>(matrix, N);
}

__global__ void z_symmetrize(hipDoubleComplex *matrix, int N)
{

	unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
	unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;

	if((xIndex < N) && (yIndex < N) && (yIndex>=xIndex)){
	    unsigned int index_1  = yIndex * N + xIndex;
	    unsigned int index_2  = xIndex * N + yIndex;
	    hipDoubleComplex val_1 = matrix[index_1];
	    hipDoubleComplex val_2 = matrix[index_2];
	    
	    matrix[index_1].x     = (val_1.x+val_2.x)/2.0;
	    matrix[index_1].y     = (val_1.y-val_2.y)/2.0;
	    matrix[index_2].x     = (val_1.x+val_2.x)/2.0;
	    matrix[index_2].y     = -(val_1.y-val_2.y)/2.0;
	}

	__syncthreads();
}

extern "C"
void z_symmetrize_matrix(CPX *matrix,int N,hipStream_t stream){

    uint i_size = N + (BLOCK_DIM-(N%BLOCK_DIM));

    dim3 grid(i_size / BLOCK_DIM, i_size / BLOCK_DIM, 1);
    dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);

    z_symmetrize<<< grid, threads, 0, stream >>>((hipDoubleComplex*)matrix, N);
}

__global__ void z_symmetrize_2(hipDoubleComplex *matrix, int N)
{

	unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
	unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;

	if((xIndex < N) && (yIndex < N) && (yIndex>=xIndex)){
	    unsigned int index_1  = yIndex * N + xIndex;
	    unsigned int index_2  = xIndex * N + yIndex;
	    hipDoubleComplex val_1 = matrix[index_1];

	    if(yIndex==xIndex){
	        matrix[index_1].x = 0.0;
		matrix[index_1].y = val_1.y;
	    }else{
	        matrix[index_2].x = -val_1.x;
		matrix[index_2].y = val_1.y;
	    }
	}

	__syncthreads();
}

extern "C"
void z_symmetrize_matrix_2(CPX *matrix,int N,hipStream_t stream){

    uint i_size = N + (BLOCK_DIM-(N%BLOCK_DIM));

    dim3 grid(i_size / BLOCK_DIM, i_size / BLOCK_DIM, 1);
    dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);

    z_symmetrize_2<<< grid, threads, 0, stream >>>((hipDoubleComplex*)matrix, N);
}
